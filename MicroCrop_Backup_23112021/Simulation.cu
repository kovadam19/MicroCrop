#include "hip/hip_runtime.h"
﻿#include "Simulation.h"


__global__ void resetNodesCUDA(Node* nodes,
                               int	 number_of_nodes)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_nodes; i += stride)
    {
        nodes[i].force = make_double3(0.0, 0.0, 0.0);
    }
}


__global__ void updateAxialSpringsCUDA(AxialSpring* axial_springs,
    int		number_of_springs,
    IntersectionNode* intersection_nodes)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_springs; i += stride)
    {
        if (axial_springs[i].status == 1)
        {
            calculateAxialSpring(axial_springs, intersection_nodes, i);
        }
    }
}

//__global__ void applyAxialSpringForcesCUDA(AxialSpring* axial_springs,
//    int		number_of_springs,
//    IntersectionNode* intersection_nodes,
//    Node* nodes)
//{
//    applyAxialSpringForces(axial_springs, number_of_springs, intersection_nodes, nodes);
//}

__global__ void updateRotationalSpringsCUDA(RotationalSpring* rotational_springs,
    int		number_of_rsprings,
    AxialSpring* axial_springs,
    IntersectionNode* intersection_nodes)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_rsprings; i += stride)
    {
        if (rotational_springs[i].status == 1)
        {
            calculateRotationalSpring(rotational_springs, axial_springs, intersection_nodes, i);
        }
    }
}

//__global__ void applyRotationalSpringForcesCUDA(RotationalSpring* rotational_springs,
//    int		number_of_rsprings,
//    AxialSpring* axial_springs,
//    IntersectionNode* intersection_nodes,
//    Node* nodes)
//{
//
//    applyRotationalSpringForces(rotational_springs, number_of_rsprings, axial_springs, intersection_nodes, nodes);
//}

//__global__ void applyExternalForcesCUDA(ExternalForce* external_forces,
//    int	number_of_forces,
//    Node* nodes,
//    double			time)
//{
//    applyExternalForces(external_forces, number_of_forces, nodes, time);
//}

__global__ void updateNodesCUDA(Node* nodes,
    int	number_of_nodes,
    double			timestep)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_nodes; i += stride)
    {
        integrateNode(nodes, i, timestep);
    }
}

__global__ void updateIntersectionNodesCUDA(IntersectionNode* intersection_nodes,
    int		number_of_inodes,
    Node* nodes)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_inodes; i += stride)
    {
        if (intersection_nodes[i].status == 1)
        {
            calculateIntersectionNodePosition(intersection_nodes, nodes, i);
            calculateIntersectionNodeVelocity(intersection_nodes, nodes, i);
        }
    }
}

__global__ void updateFacesCUDA(Face* faces,
    int	number_of_faces,
    Node* nodes)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_faces; i += stride)
    {
        if (faces[i].status == 1)
        {
            calculateFaceCenter(faces, nodes, i);
            calculateFaceNormal(faces, nodes, i);
            calculateFaceArea(faces, nodes, i);
        }
    }
}

__global__ void updateCellsCUDA(Cell* cells,
    int	number_of_cells,
    Node* nodes,
    IntersectionNode* intersection_nodes,
    Face* faces,
    AxialSpring* axial_springs,
    RotationalSpring* rotational_springs)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < number_of_cells; i += stride)
    {
        if (cells[i].status == 1)
        {
            calculateCellCenter(cells, nodes, i);
            calculateCellVolume(cells, nodes, i);
            calculateCellCircumsphere(cells, nodes, i);
            checkCellDamage(cells, i, faces, intersection_nodes, axial_springs, rotational_springs);
        }
    }
}



__host__ hipError_t runSimulationCUDA(CellContainer& host_cells,
    FaceContainer& host_faces,
    NodeContainer& host_nodes,
    IntersectionNodeContainer& host_intersection_nodes,
    AxialSpringContainer& host_axial_springs,
    RotationalSpringContainer& host_rotational_springs,
    ExternalForceContainer& host_external_forces,
    Settings& host_settings)
{
    // Variables for testing performance
    auto start = std::chrono::high_resolution_clock::now();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    
    std::cout << "Initializing the GPU device..." << std::endl;

    // Initializing the device variables
    Cell* dev_cells;
    const int number_of_cells = host_cells.size();
    Face* dev_faces;
    const int number_of_faces = host_faces.size();
    Node* dev_nodes;
    const int number_of_nodes = host_nodes.size();
    IntersectionNode* dev_intersection_nodes;
    const int number_of_intersection_nodes = host_intersection_nodes.size();
    AxialSpring* dev_axial_springs;
    const int number_of_axial_springs = host_axial_springs.size();
    RotationalSpring* dev_rotational_springs;
    const int number_of_rotational_springs = host_rotational_springs.size();
    //ExternalForce* dev_external_forces;
    //const int number_of_external_forces = host_external_forces.size();
    hipError_t cudaStatus;

    // Initialize the device
    cudaStatus = hipSetDevice(host_settings.GPU_device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    // Allocate GPU buffers for device containers
    cudaStatus = hipMalloc((void**)&dev_cells, number_of_cells * sizeof(Cell));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_cells!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_faces, number_of_faces * sizeof(Face));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_faces!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nodes, number_of_nodes * sizeof(Node));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_nodes!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_intersection_nodes, number_of_intersection_nodes * sizeof(IntersectionNode));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_intersection_nodes!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_axial_springs, number_of_axial_springs * sizeof(AxialSpring));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_axial_springs!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rotational_springs, number_of_rotational_springs * sizeof(RotationalSpring));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_rotational_springs!");
        goto Error;
    }

    //cudaStatus = hipMalloc((void**)&dev_external_forces, number_of_external_forces * sizeof(ExternalForce));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed on dev_external_forces!");
    //    goto Error;
    //}

    // Copy containers from host memory to GPU buffers
    cudaStatus = hipMemcpy(dev_cells, &host_cells[0], number_of_cells * sizeof(Cell), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_cells!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_faces, &host_faces[0], number_of_faces * sizeof(Face), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_faces!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_nodes, &host_nodes[0], number_of_nodes * sizeof(Node), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_nodes!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_intersection_nodes, &host_intersection_nodes[0], number_of_intersection_nodes * sizeof(IntersectionNode), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_intersection_nodes!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_axial_springs, &host_axial_springs[0], number_of_axial_springs * sizeof(AxialSpring), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_axial_springs!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rotational_springs, &host_rotational_springs[0], number_of_rotational_springs * sizeof(RotationalSpring), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_rotational_springs!");
        goto Error;
    }

    //cudaStatus = hipMemcpy(dev_external_forces, &host_external_forces[0], number_of_external_forces * sizeof(ExternalForce), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed on dev_external_forces!");
    //    goto Error;
    //}

    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Execution time of GPU initialization in milliseconds: " << duration.count() << std::endl;


    std::cout << "Starting calculation on the GPU device..." << std::endl;
    start = std::chrono::high_resolution_clock::now();


    double simulation_time = host_settings.start_time;
    double simulation_end_time = host_settings.end_time;
    int save_interval = int(host_settings.save_interval / host_settings.timestep);
    int step_counter = 0;
    int export_counter = 0;

    int threads_per_block = host_settings.GPU_threads_per_block;
    int number_of_blocks = host_settings.GPU_number_of_blocks;


    while (simulation_time <= simulation_end_time)
    {
        if (step_counter == save_interval)
        {
            if (simulation_time != host_settings.start_time)
            {
                stop = std::chrono::high_resolution_clock::now();
                duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
                std::cout << "Execution time of " << save_interval << " iterations in milliseconds: " << duration.count() << std::endl;
            }

            std::cout << "Simulation is " << (simulation_time / simulation_end_time) * 100 << " % completed." << std::endl;

            // Saving
            start = std::chrono::high_resolution_clock::now();
            std::cout << "Saving at time " << simulation_time << " second." << std::endl;

            // Copy containers from GPU buffer to host memory
            if (host_settings.save_nodes == 1 || host_settings.save_faces == 1 || host_settings.save_cells == 1)
            {
                cudaStatus = hipMemcpy(&host_nodes[0], dev_nodes, number_of_nodes * sizeof(Node), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed on host_nodes!");
                    goto Error;
                }
            }
    
            if (host_settings.save_faces == 1)
            {
                cudaStatus = hipMemcpy(&host_faces[0], dev_faces, number_of_faces * sizeof(Face), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed on host_faces!");
                    goto Error;
                }
            }

            if (host_settings.save_cells == 1)
            {
                cudaStatus = hipMemcpy(&host_cells[0], dev_cells, number_of_cells * sizeof(Cell), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed on host_cells!");
                    goto Error;
                }
            }

            if (host_settings.save_axial_springs == 1 || host_settings.save_rotational_springs == 1)
            {
                cudaStatus = hipMemcpy(&host_intersection_nodes[0], dev_intersection_nodes, number_of_intersection_nodes * sizeof(IntersectionNode), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed on host_intersection_nodes!");
                    goto Error;
                }
            }

            if (host_settings.save_axial_springs == 1 || host_settings.save_rotational_springs == 1)
            {
                cudaStatus = hipMemcpy(&host_axial_springs[0], dev_axial_springs, number_of_axial_springs * sizeof(AxialSpring), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed on host_axial_springs!");
                    goto Error;
                }
            }

            if (host_settings.save_rotational_springs == 1)
            {
                cudaStatus = hipMemcpy(&host_rotational_springs[0], dev_rotational_springs, number_of_rotational_springs * sizeof(RotationalSpring), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed on host_rotational_springs!");
                    goto Error;
                }
            }

            exportSimulationData(host_cells,
                host_faces,
                host_nodes,
                host_intersection_nodes,
                host_axial_springs,
                host_rotational_springs,
                host_settings,
                export_counter);

            export_counter++;
            step_counter = 0;

            stop = std::chrono::high_resolution_clock::now();
            duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
            std::cout << "Execution time of the save in milliseconds: " << duration.count() << std::endl;

            start = std::chrono::high_resolution_clock::now();
        }







        start = std::chrono::high_resolution_clock::now();
        resetNodesCUDA<<<number_of_blocks, threads_per_block>>>(dev_nodes, number_of_nodes);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "resetNodesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching resetNodesCUDA!\n", cudaStatus);
            goto Error;
        }

        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of resetNodesCUDA in milliseconds: " << duration.count() << std::endl;







        start = std::chrono::high_resolution_clock::now();
        updateAxialSpringsCUDA<<<number_of_blocks, threads_per_block >>>(dev_axial_springs, number_of_axial_springs, dev_intersection_nodes);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updateAxialSpringsCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateAxialSpringsCUDA!\n", cudaStatus);
            goto Error;
        }

        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of updateAxialSpringsCUDA in milliseconds: " << duration.count() << std::endl;







        //start = std::chrono::high_resolution_clock::now();
        //applyAxialSpringForcesCUDA<<<1, 1>>>(dev_axial_springs, number_of_axial_springs, dev_intersection_nodes, dev_nodes);

        //// Check for any errors launching the kernel
        //cudaStatus = hipGetLastError();
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "applyAxialSpringForcesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
        //    goto Error;
        //}

        //// Waiting for the kernel to finish
        //cudaStatus = hipDeviceSynchronize();
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching applyAxialSpringForcesCUDA!\n", cudaStatus);
        //    goto Error;
        //}

        //stop = std::chrono::high_resolution_clock::now();
        //duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        //std::cout << "Execution time of applyAxialSpringForcesCUDA in milliseconds: " << duration.count() << std::endl;








        start = std::chrono::high_resolution_clock::now();
        updateRotationalSpringsCUDA<<<number_of_blocks, threads_per_block >>>(dev_rotational_springs, number_of_rotational_springs, dev_axial_springs, dev_intersection_nodes);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updateRotationalSpringsCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateRotationalSpringsCUDA!\n", cudaStatus);
            goto Error;
        }

        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of updateRotationalSpringsCUDA in milliseconds: " << duration.count() << std::endl;






        //start = std::chrono::high_resolution_clock::now();
        //applyRotationalSpringForcesCUDA<<<1, 1 >>>(dev_rotational_springs, number_of_rotational_springs, dev_axial_springs, dev_intersection_nodes, dev_nodes);

        //// Check for any errors launching the kernel
        //cudaStatus = hipGetLastError();
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "applyRotationalSpringForcesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
        //    goto Error;
        //}

        //// Waiting for the kernel to finish
        //cudaStatus = hipDeviceSynchronize();
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching applyRotationalSpringForcesCUDA!\n", cudaStatus);
        //    goto Error;
        //}

        //stop = std::chrono::high_resolution_clock::now();
        //duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        //std::cout << "Execution time of applyRotationalSpringForcesCUDA in milliseconds: " << duration.count() << std::endl;






        //start = std::chrono::high_resolution_clock::now();
        //applyExternalForcesCUDA<<<1, 1 >>>(dev_external_forces, number_of_external_forces, dev_nodes, simulation_time);

        //// Check for any errors launching the kernel
        //cudaStatus = hipGetLastError();
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "applyExternalForcesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
        //    goto Error;
        //}

        //// Waiting for the kernel to finish
        //cudaStatus = hipDeviceSynchronize();
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching applyExternalForcesCUDA!\n", cudaStatus);
        //    goto Error;
        //}

        //stop = std::chrono::high_resolution_clock::now();
        //duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        //std::cout << "Execution time of applyExternalForcesCUDA in milliseconds: " << duration.count() << std::endl;

        start = std::chrono::high_resolution_clock::now();
        cudaStatus = hipMemcpy(&host_nodes[0], dev_nodes, number_of_nodes * sizeof(Node), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on host_nodes!");
            goto Error;
        }

        cudaStatus = hipMemcpy(&host_intersection_nodes[0], dev_intersection_nodes, number_of_intersection_nodes * sizeof(IntersectionNode), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on host_intersection_nodes!");
            goto Error;
        }
       
        cudaStatus = hipMemcpy(&host_axial_springs[0], dev_axial_springs, number_of_axial_springs * sizeof(AxialSpring), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on host_axial_springs!");
            goto Error;
        }
        
        cudaStatus = hipMemcpy(&host_rotational_springs[0], dev_rotational_springs, number_of_rotational_springs * sizeof(RotationalSpring), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on host_rotational_springs!");
            goto Error;
        }
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of copying data from device to host to apply forces in milliseconds: " << duration.count() << std::endl;

        start = std::chrono::high_resolution_clock::now();
        applyAxialSpringForces(&host_axial_springs[0],
            number_of_axial_springs,
            &host_intersection_nodes[0],
            &host_nodes[0]);
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of applyAxialSpringForces in milliseconds: " << duration.count() << std::endl;

        start = std::chrono::high_resolution_clock::now();
        applyRotationalSpringForces(&host_rotational_springs[0],
            number_of_rotational_springs,
            &host_axial_springs[0],
            &host_intersection_nodes[0],
            &host_nodes[0]);
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of applyRotationalSpringForces in milliseconds: " << duration.count() << std::endl;

        start = std::chrono::high_resolution_clock::now();
        applyExternalForces(&host_external_forces[0],
            host_external_forces.size(),
            &host_nodes[0],
            simulation_time);
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of applyExternalForces in milliseconds: " << duration.count() << std::endl;

        start = std::chrono::high_resolution_clock::now();
        cudaStatus = hipMemcpy(dev_nodes, &host_nodes[0], number_of_nodes * sizeof(Node), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on dev_nodes!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_intersection_nodes, &host_intersection_nodes[0], number_of_intersection_nodes * sizeof(IntersectionNode), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on dev_intersection_nodes!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_axial_springs, &host_axial_springs[0], number_of_axial_springs * sizeof(AxialSpring), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on dev_axial_springs!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_rotational_springs, &host_rotational_springs[0], number_of_rotational_springs * sizeof(RotationalSpring), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed on dev_rotational_springs!");
            goto Error;
        }
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of copying data from host to device after applying forces in milliseconds: " << duration.count() << std::endl;



        start = std::chrono::high_resolution_clock::now();
        updateNodesCUDA <<<number_of_blocks, threads_per_block >>> (dev_nodes, number_of_nodes, host_settings.timestep);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updateNodesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateNodesCUDA!\n", cudaStatus);
            goto Error;
        }
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of updateNodesCUDA in milliseconds: " << duration.count() << std::endl;







        start = std::chrono::high_resolution_clock::now();
        updateIntersectionNodesCUDA <<<number_of_blocks, threads_per_block >>> (dev_intersection_nodes, number_of_intersection_nodes, dev_nodes);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updateIntersectionNodesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateIntersectionNodesCUDA!\n", cudaStatus);
            goto Error;
        }

        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of updateIntersectionNodesCUDA in milliseconds: " << duration.count() << std::endl;





        start = std::chrono::high_resolution_clock::now();
        updateFacesCUDA <<<number_of_blocks, threads_per_block >>> (dev_faces, number_of_faces, dev_nodes);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updateFacesCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateFacesCUDA!\n", cudaStatus);
            goto Error;
        }

        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of updateFacesCUDA in milliseconds: " << duration.count() << std::endl;






        start = std::chrono::high_resolution_clock::now();
        updateCellsCUDA <<<number_of_blocks, threads_per_block >>> (dev_cells, number_of_cells, dev_nodes, dev_intersection_nodes, dev_faces, dev_axial_springs, dev_rotational_springs);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updateCellsCUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Waiting for the kernel to finish
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateCellsCUDA!\n", cudaStatus);
            goto Error;
        }

        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        std::cout << "Execution time of updateCellsCUDA in milliseconds: " << duration.count() << std::endl;




        step_counter++;
        simulation_time += host_settings.timestep;
    }


Error:
    // Free the GPU memory
    hipFree(dev_cells);
    hipFree(dev_faces);
    hipFree(dev_nodes);
    hipFree(dev_intersection_nodes);
    hipFree(dev_axial_springs);
    hipFree(dev_rotational_springs);
    //hipFree(dev_external_forces);

    return cudaStatus;
}



__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
